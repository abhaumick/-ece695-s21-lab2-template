#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, int scale, int size) {
	//	TODO: replaced with full 3D thread block computations
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId < size) {
		y[threadId] = scale * x[threadId] + y[threadId];
	}
}

int runGpuSaxpy(int vectorSize) {

	uint64_t vectorBytes = vectorSize * sizeof(float);
	
	printf("Hello GPU Saxpy!\n");
	std::srand(std::time(0));
	
	float * h_x, * h_y, * h_z;

	h_x = (float *) malloc(vectorSize * sizeof(float));
	h_y = (float *) malloc(vectorSize * sizeof(float));
	h_z = (float *) malloc(vectorSize * sizeof(float));

	if (h_x == NULL || h_y == NULL || h_z == NULL) {
		std::cerr << "Unable to malloc memory ... Exiting!\n";
		exit(0);
	}
	
	vectorInit(h_x, vectorSize);
	vectorInit(h_y, vectorSize);
	float scale = 2.0f;
	
	float * d_x, * d_y;

	hipDeviceReset();
	hipError_t cudaStatus;
	int deviceCount = 0;

	cudaStatus = hipGetDeviceCount ( &deviceCount ); 
	if (deviceCount == 0) {
		std::cerr << "No CUDA Devices found!\n";
		return 0;
	}
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "hipSetDevice failed!\n";
		return 0;
	}

	cudaStatus = hipMalloc((void**)&d_x, vectorSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		std::cerr << "hipMalloc failed!\n";
		return 0;
	}
	cudaStatus = hipMalloc((void**)&d_y, vectorSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		std::cerr << "hipMalloc failed!\n";
		return 0;
	}

	//printVector(h_x, vectorSize);
	//printVector(h_y, vectorSize);

	hipMemcpy(d_x, h_x, vectorBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, vectorBytes, hipMemcpyHostToDevice);

	int threadDim = 1024;
	int blockDim = ((vectorSize - 1) / threadDim) + 1;

	std::cout << "Launching <<< " << blockDim << ", " << threadDim << " >>> kernel\n";

	saxpy_gpu <<< blockDim, threadDim >>> (d_x, d_y, scale, vectorSize);
	hipDeviceSynchronize();

	hipMemcpy(h_z, d_y, vectorBytes, hipMemcpyDeviceToHost);

	int errorCount = verifyVector(h_x, h_y, h_z, scale, vectorSize);

	std::cout << "Found " << errorCount << " / " << vectorSize << " errors in vector! \n";

	hipFree(d_x);
	hipFree(d_y);

	free(h_x);
	free(h_y);
	free(h_z);
	return 0;
}

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	// Find unique thread id
	uint64_t threadId = threadIdx.x + blockIdx.x * blockDim.x;

	// Setup RNG
	hiprandState_t rng;
	hiprand_init(clock64(), threadId, 0, &rng);

	// Init counters
	uint64_t hitCount = 0;
	float x = 0.0f, y = 0.0f;

	if (threadId < pSumSize) {
		// Generate points & compute probability
		for (uint64_t iter = 0; iter < sampleSize; ++iter) {
			x = hiprand_uniform(&rng);
			y = hiprand_uniform(&rng);
			if ( int(x * x + y * y) == 0 ) {
				++ hitCount;
			}
		}

		//  Write out results to memory
		pSums[threadId] = hitCount;
	}
}

/**
* @brief Optional GPU kernel to reduce a set of partial sums into a smaller set
*			by summing a subset into a single value
* 
* @param pSums 
* @param totals 
* @param pSumSize 
* @param reduceSize 
* @return void 
*/
__global__ void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	uint64_t tSum = 0;
	uint64_t threadId = threadIdx.x + blockIdx.x * blockDim.x;
	uint64_t arrayOffset = threadId * reduceSize;

	for (uint64_t idx = 0; idx < reduceSize; ++ idx) {
		if (arrayOffset + idx < pSumSize)
			tSum += pSums[arrayOffset + idx];
	}

	totals[threadId] = tSum;
}

/**
 * @brief Entrypoint for GPU Monte-Carlo estimation of Pi
 * 
 * @param generateThreadCount 	uint64_t	total number of generate threads	
 * @param sampleSize 			uint64_t	sample of points evaluated by each thread
 * @param reduceThreadCount 	uint64_t	number of reduction threads
 * @param reduceSize 			uint64_t	number of pSums summed by each reduce thread
 * @return int 
 */
int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}


/**
 * @brief main body for Monte-Carlo Pi estimation
 * 
 * @param generateThreadCount 	uint64_t	total number of generate threads	
 * @param sampleSize 			uint64_t	sample of points evaluated by each thread
 * @param reduceThreadCount 	uint64_t	number of reduction threads
 * @param reduceSize 			uint64_t	number of pSums summed by each reduce thread
 * @return double 	approx value of pi
 */
double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	uint64_t * hTSums, * hPSums;
	uint64_t * dPSums, * dTSums;

	reduceThreadCount = std::ceil(generateThreadCount / reduceSize);
	
	hPSums = new uint64_t[generateThreadCount];
	hTSums = new uint64_t[reduceThreadCount];

	//	Get CUDA Device Details
	int deviceId;
	hipDeviceProp_t deviceProp;
	hipGetDevice(&deviceId);
	hipGetDeviceProperties (&deviceProp, deviceId);

	hipDeviceReset();
	hipMalloc(&dPSums, generateThreadCount * sizeof(uint64_t));
	hipMalloc(&dTSums, reduceThreadCount * sizeof(uint64_t));


	uint64_t blockDim = std::min(generateThreadCount, (uint64_t)deviceProp.maxThreadsPerBlock);
	uint64_t gridDim = ((generateThreadCount - 1) / deviceProp.maxThreadsPerBlock) + 1;
	
	#ifndef DEBUG_PRINT_DISABLE
		printf("Launching kernel <<< %d, %d >>> \n", gridDim, blockDim);
	#endif

	generatePoints <<<gridDim, blockDim>>> (dPSums, generateThreadCount, sampleSize);
	
	gpuErrchk( hipPeekAtLastError() );

	blockDim = std::min(reduceThreadCount, (uint64_t)deviceProp.maxThreadsPerBlock);
	gridDim = ((reduceThreadCount - 1) / deviceProp.maxThreadsPerBlock) + 1;
	printf("Launching kernel <<< %d, %d >>> \n", gridDim, blockDim);
	reduceCounts <<<gridDim, blockDim>>> (dPSums, dTSums, generateThreadCount, reduceSize);

	hipMemcpy(hPSums, dPSums, generateThreadCount * sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipMemcpy(hTSums, dTSums, reduceThreadCount * sizeof(uint64_t), hipMemcpyDeviceToHost);
	gpuErrchk( hipPeekAtLastError() );

	hipFree(dTSums);
	hipFree(dPSums);
	
	uint64_t totalHitCount = 0;
	for (int idx = 0; idx < reduceThreadCount; ++idx) {
		//	Each count is #hits out of reduceSize * sampleSize
		totalHitCount += hTSums[idx];
	}
	std::cout << std::setprecision(10);
	//std::cout << "Total Hits = " << totalHitCount << " / " << (generateThreadCount * sampleSize) << " \n";

	double approxPi = ( ((double)totalHitCount / generateThreadCount) / sampleSize );
	// Adjust for quarter circle
	approxPi = approxPi * 4.0;
	return approxPi;
}
